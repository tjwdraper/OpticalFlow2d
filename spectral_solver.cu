#include "hip/hip_runtime.h"
#include <spectral_solver.cuh>
#include <mex.h>

#define PI 3.14159265

__global__ void getG (coord3D *g, coord3D *velocity, coord3D *force, float gamma, dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    // Calculate new entry
    g[idx] = velocity[idx] - force[idx] * gamma;
    // Done
    return;
}

__global__ void getGcoefs (hipfftComplex *coefs_g_x, hipfftComplex *coefs_g_y, hipfftComplex *coefs_g_z,
                           hipfftComplex *coefs_v_x, hipfftComplex *coefs_v_y, hipfftComplex *coefs_v_z,
                           hipfftComplex *coefs_f_x, hipfftComplex *coefs_f_y, hipfftComplex *coefs_f_z,
                           float gamma,
                           dim dimin) {
    /// Determine the relative index of the current GPU thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use
    idx = k + j*dimin.z + i*dimin.z*dimin.y;
    // Get update
    coefs_g_x[idx].x = coefs_v_x[idx].x - coefs_f_x[idx].x * gamma;
    coefs_g_x[idx].y = coefs_v_x[idx].y - coefs_f_x[idx].y * gamma;
    coefs_g_y[idx].x = coefs_v_y[idx].x - coefs_f_y[idx].x * gamma;
    coefs_g_y[idx].y = coefs_v_y[idx].y - coefs_f_y[idx].y * gamma;
    coefs_g_z[idx].x = coefs_v_z[idx].x - coefs_f_z[idx].x * gamma;
    coefs_g_z[idx].y = coefs_v_z[idx].y - coefs_f_z[idx].y * gamma;
    //
    return;
}

__global__ void getComponentsKernel(hipfftReal *g_x, hipfftReal *g_y, hipfftReal *g_z, coord3D *g, dim dimin) {
    /// Determine the relative index of the current GPU thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use
    int idxColMajor = i + j*dimin.x + k*dimin.x*dimin.y;
    int idxRowMajor = k + j*dimin.z + i*dimin.z*dimin.y;
    // Get update
    g_x[idxRowMajor] = g[idxColMajor].x;
    g_y[idxRowMajor] = g[idxColMajor].y;
    g_z[idxRowMajor] = g[idxColMajor].z;
}

__host__ void getComponents(hipfftReal *g_x, hipfftReal *g_y, hipfftReal *g_z, coord3D *g, dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);

    // Decompose the components of g
    getComponentsKernel <<<nBlocks, threadsPerBlock>>> (g_x, g_y, g_z, g, dimin);

    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch getComponents kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    //Done
    return;
}

__global__ void divide(hipfftComplex *coefs_g_x, hipfftComplex *coefs_g_y, hipfftComplex *coefs_g_z, 
    float *eigenvals, float mu, float lambda, float gamma, dim dimin) {
    /// Determine the relative index of the current GPU thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use
    idx = k + j*dimin.z + i*dimin.z*dimin.y;

    // Real part
    const int step = dimin.x*dimin.y*dimin.z;
    hipfftComplex a,b,c;
    a.x = eigenvals[idx + 0*step] * coefs_g_x[idx].x + eigenvals[idx + 3*step] * coefs_g_y[idx].x + eigenvals[idx + 4*step] * coefs_g_z[idx].x;
    b.x = eigenvals[idx + 3*step] * coefs_g_x[idx].x + eigenvals[idx + 1*step] * coefs_g_y[idx].x + eigenvals[idx + 5*step] * coefs_g_z[idx].x;
    c.x = eigenvals[idx + 4*step] * coefs_g_x[idx].x + eigenvals[idx + 5*step] * coefs_g_y[idx].x + eigenvals[idx + 2*step] * coefs_g_z[idx].x;

    // Imaginary part
    a.y = eigenvals[idx + 0*step] * coefs_g_x[idx].y + eigenvals[idx + 3*step] * coefs_g_y[idx].y + eigenvals[idx + 4*step] * coefs_g_z[idx].y;
    b.y = eigenvals[idx + 3*step] * coefs_g_x[idx].y + eigenvals[idx + 1*step] * coefs_g_y[idx].y + eigenvals[idx + 5*step] * coefs_g_z[idx].y;
    c.y = eigenvals[idx + 4*step] * coefs_g_x[idx].y + eigenvals[idx + 5*step] * coefs_g_y[idx].y + eigenvals[idx + 2*step] * coefs_g_z[idx].y;

    // Set new values
    coefs_g_x[idx] = a;
    coefs_g_y[idx] = b;
    coefs_g_z[idx] = c;
}

__global__ void divide(hipfftComplex *coefs_v_x, hipfftComplex *coefs_v_y, hipfftComplex *coefs_v_z,
    hipfftComplex *coefs_g_x, hipfftComplex *coefs_g_y, hipfftComplex *coefs_g_z, 
    float *eigenvals, float mu, float lambda, float gamma, dim dimin) {
    /// Determine the relative index of the current GPU thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use
    idx = k + j*dimin.z + i*dimin.z*dimin.y;

    // Real part
    const int step = dimin.x*dimin.y*dimin.z;
    coefs_v_x[idx].x = eigenvals[idx + 0*step] * coefs_g_x[idx].x + eigenvals[idx + 3*step] * coefs_g_y[idx].x + eigenvals[idx + 4*step] * coefs_g_z[idx].x;
    coefs_v_y[idx].x = eigenvals[idx + 3*step] * coefs_g_x[idx].x + eigenvals[idx + 1*step] * coefs_g_y[idx].x + eigenvals[idx + 5*step] * coefs_g_z[idx].x;
    coefs_v_z[idx].x = eigenvals[idx + 4*step] * coefs_g_x[idx].x + eigenvals[idx + 5*step] * coefs_g_y[idx].x + eigenvals[idx + 2*step] * coefs_g_z[idx].x;

    // Imaginary part
    coefs_v_x[idx].y = eigenvals[idx + 0*step] * coefs_g_x[idx].y + eigenvals[idx + 3*step] * coefs_g_y[idx].y + eigenvals[idx + 4*step] * coefs_g_z[idx].y;
    coefs_v_y[idx].y = eigenvals[idx + 3*step] * coefs_g_x[idx].y + eigenvals[idx + 1*step] * coefs_g_y[idx].y + eigenvals[idx + 5*step] * coefs_g_z[idx].y;
    coefs_v_z[idx].y = eigenvals[idx + 4*step] * coefs_g_x[idx].y + eigenvals[idx + 5*step] * coefs_g_y[idx].y + eigenvals[idx + 2*step] * coefs_g_z[idx].y;
}

__global__ void construct(coord3D *g, hipfftReal *g_x, hipfftReal *g_y, hipfftReal *g_z, dim dimin) {
    /// Determine the relative index of the current GPU thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Get the length of the array
    int length = dimin.x * dimin.y * dimin.z;
    // Compute the absolute index for future use
    int idxColMajor = i + j*dimin.x + k*dimin.x*dimin.y;
    int idxRowMajor = k + j*dimin.z + i*dimin.z*dimin.y;
    // Get update
    g[idxColMajor].x = g_x[idxRowMajor]/length;
    g[idxColMajor].y = g_y[idxRowMajor]/length;
    g[idxColMajor].z = g_z[idxRowMajor]/length;
}

__global__ void divide_v2(
    hipfftComplex* coefs_v_x, hipfftComplex* coefs_v_y, hipfftComplex* coefs_v_z,
    hipfftComplex* coefs_f_x, hipfftComplex* coefs_f_y, hipfftComplex* coefs_f_z,
    float* eigenvals,
    float mu, float lambda, float gamma,
    dim dimin,
    int siter) {
    // Determine the relative index of the current GPU thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    
    // Check that the indices are within image bounds
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    
    // Compute the absolute index for future use
    idx = k + j * dimin.z + i * dimin.z * dimin.y;
    
    // Get step
    const int step = dimin.x * dimin.y * dimin.z;
    
    // Get components of inverse matrix
    float Axx = eigenvals[idx + 0 * step];
    float Ayy = eigenvals[idx + 1 * step];
    float Azz = eigenvals[idx + 2 * step];
    float Axy = eigenvals[idx + 3 * step];
    float Axz = eigenvals[idx + 4 * step];
    float Ayz = eigenvals[idx + 5 * step];
    
    hipfftComplex coefs_v_x_val = coefs_v_x[idx];
    hipfftComplex coefs_v_y_val = coefs_v_y[idx];
    hipfftComplex coefs_v_z_val = coefs_v_z[idx];
    
    hipfftComplex coefs_f_x_val = coefs_f_x[idx];
    hipfftComplex coefs_f_y_val = coefs_f_y[idx];
    hipfftComplex coefs_f_z_val = coefs_f_z[idx];
    
    float coefs_v_x_real = coefs_v_x_val.x;
    float coefs_v_x_imag = coefs_v_x_val.y;
    
    float coefs_v_y_real = coefs_v_y_val.x;
    float coefs_v_y_imag = coefs_v_y_val.y;
    
    float coefs_v_z_real = coefs_v_z_val.x;
    float coefs_v_z_imag = coefs_v_z_val.y;
    
    float coefs_f_x_real = coefs_f_x_val.x;
    float coefs_f_x_imag = coefs_f_x_val.y;
    
    float coefs_f_y_real = coefs_f_y_val.x;
    float coefs_f_y_imag = coefs_f_y_val.y;
    
    float coefs_f_z_real = coefs_f_z_val.x;
    float coefs_f_z_imag = coefs_f_z_val.y;
    
    for (int r = 0; r < siter; ++r) {
        // Real part
        coefs_v_x_real = Axx * (coefs_v_x_real - coefs_f_x_real * gamma) +
                         Axy * (coefs_v_y_real - coefs_f_y_real * gamma) +
                         Axz * (coefs_v_z_real - coefs_f_z_real * gamma);
        
        coefs_v_y_real = Axy * (coefs_v_x_real - coefs_f_x_real * gamma) +
                         Ayy * (coefs_v_y_real - coefs_f_y_real * gamma) +
                         Ayz * (coefs_v_z_real - coefs_f_z_real * gamma);
        
        coefs_v_z_real = Axz * (coefs_v_x_real - coefs_f_x_real * gamma) +
                         Ayz * (coefs_v_y_real - coefs_f_y_real * gamma) +
                         Azz * (coefs_v_z_real - coefs_f_z_real * gamma);
        
        // Imaginary part
        coefs_v_x_imag = Axx * (coefs_v_x_imag - coefs_f_x_imag * gamma) +
                         Axy * (coefs_v_y_imag - coefs_f_y_imag * gamma) +
                         Axz * (coefs_v_z_imag - coefs_f_z_imag * gamma);
        
        coefs_v_y_imag = Axy * (coefs_v_x_imag - coefs_f_x_imag * gamma) +
                         Ayy * (coefs_v_y_imag - coefs_f_y_imag * gamma) +
                         Ayz * (coefs_v_z_imag - coefs_f_z_imag * gamma);
        
        coefs_v_z_imag = Axz * (coefs_v_x_imag - coefs_f_x_imag * gamma) +
                         Ayz * (coefs_v_y_imag - coefs_f_y_imag * gamma) +
                         Azz * (coefs_v_z_imag - coefs_f_z_imag * gamma);
    }
    
    // Update the output values
    coefs_v_x[idx].x = coefs_v_x_real;
    coefs_v_x[idx].y = coefs_v_x_imag;
    
    coefs_v_y[idx].x = coefs_v_y_real;
    coefs_v_y[idx].y = coefs_v_y_imag;
    
    coefs_v_z[idx].x = coefs_v_z_real;
    coefs_v_z[idx].y = coefs_v_z_imag;
}


__host__ void spectral_update_v2(coord3D *velocity,
    hipfftReal *g_x, hipfftReal *g_y, hipfftReal *g_z,
    hipfftComplex *coefs_f_x, hipfftComplex *coefs_f_y, hipfftComplex *coefs_f_z,
    hipfftComplex *coefs_v_x, hipfftComplex *coefs_v_y, hipfftComplex *coefs_v_z,
    float *eigenvals, 
    float mu, float lambda, float gamma, 
    int siter,
    dim dimin,
    hipfftHandle planbackward) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);

    // Get the dimension of the FFT
    dim dimcoefs(dimin.x, dimin.y, static_cast<int>(floor(dimin.z/2) + 1)  );

    divide_v2 <<< nBlocks, threadsPerBlock >>> (coefs_v_x, coefs_v_y, coefs_v_z,
                                                coefs_f_x, coefs_f_y, coefs_f_z,
                                                eigenvals, 
                                                mu, lambda, gamma,
                                                dimcoefs,
                                                siter);
    hipDeviceSynchronize();

    // Inverse FFT
    hipfftExecC2R(planbackward, coefs_v_x, g_x);
    hipfftExecC2R(planbackward, coefs_v_y, g_y);
    hipfftExecC2R(planbackward, coefs_v_z, g_z);

    // Compose coord3D object
    construct <<<nBlocks, threadsPerBlock>>> (velocity, g_x, g_y, g_z, dimin);

    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch update kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    //Done
    return;
}

__host__ void spectral_update(coord3D *velocity, coord3D *force, coord3D *g,
    hipfftReal *g_x, hipfftReal *g_y, hipfftReal *g_z,
    hipfftComplex *coefs_g_x, hipfftComplex *coefs_g_y, hipfftComplex *coefs_g_z,
    hipfftHandle planforward, hipfftHandle planbackward,
    float *eigenvals, 
    float mu, float lambda, float gamma, 
    int siter,
    dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);

    // Get the dimension of the FFT
    dim dimcoefs(dimin.x, dimin.y, static_cast<int>(floor(dimin.z/2) + 1)  );

    for (int r = 0; r < siter; r++) {

        // Launch kernel to get g
        getG <<< nBlocks, threadsPerBlock >>> (g, velocity, force, gamma, dimin);

        // Decompose the components of g
        getComponentsKernel <<<nBlocks, threadsPerBlock>>> (g_x, g_y, g_z, g, dimin);

        // Calculate the FFT of the individual components
        hipfftExecR2C(planforward, g_x, coefs_g_x);
        hipfftExecR2C(planforward, g_y, coefs_g_y);
        hipfftExecR2C(planforward, g_z, coefs_g_z);

        // Divide with eigenvalues of differential operator
        divide <<<nBlocks, threadsPerBlock>>> (coefs_g_x, coefs_g_y, coefs_g_z, eigenvals, mu, lambda, gamma, dimcoefs);

        // Inverse FFT
        hipfftExecC2R(planbackward, coefs_g_x, g_x);
        hipfftExecC2R(planbackward, coefs_g_y, g_y);
        hipfftExecC2R(planbackward, coefs_g_z, g_z);

        // Compose coord3D object
        construct <<<nBlocks, threadsPerBlock>>> (velocity, g_x, g_y, g_z, dimin);

        // Synchronize
        hipDeviceSynchronize();
    }

    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch update kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    //Done
    return;
}

__global__ void getEigenValsKernel(float *eigenvals, float gamma, float mu, float lambda, dim dimcoefs, dim dimin) {
    /// Determine the relative index of the current GPU thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int r = blockIdx.y;
    int q = idx / dimcoefs.x;
    int p = idx % dimcoefs.x;
    // Check that the indices are within image bounds
    if ((r < 0) || (q < 0) || (p < 0) ||
        (r > dimcoefs.z - 1) || (q > dimcoefs.y - 1) || (p > dimcoefs.x - 1)) {
        return;
    }
    // Compute the absolute index for future use
    idx = r + q*dimcoefs.z + p*dimcoefs.z*dimcoefs.y;
    // Set beta
    float beta = gamma*(mu+lambda);
    // Set the components of the diagonal matrix
    float d11 = 1.0 / (1.0 + gamma*mu*(6-2*cosf(2*PI*p/dimin.x)-2*cosf(2*PI*q/dimin.y)-2*cosf(2*PI*r/dimin.z)) + gamma*(mu+lambda)*(2-2*cosf(2*PI*p/dimin.x)+sinf(2*PI*p/dimin.x)*sinf(2*PI*p/dimin.x)));
    float d22 = 1.0 / (1.0 + gamma*mu*(6-2*cosf(2*PI*p/dimin.x)-2*cosf(2*PI*q/dimin.y)-2*cosf(2*PI*r/dimin.z)) + gamma*(mu+lambda)*(2-2*cosf(2*PI*q/dimin.y)+sinf(2*PI*q/dimin.y)*sinf(2*PI*q/dimin.y)));
    float d33 = 1.0 / (1.0 + gamma*mu*(6-2*cosf(2*PI*p/dimin.x)-2*cosf(2*PI*q/dimin.y)-2*cosf(2*PI*r/dimin.z)) + gamma*(mu+lambda)*(2-2*cosf(2*PI*r/dimin.z)+sinf(2*PI*r/dimin.z)*sinf(2*PI*r/dimin.z)));
    // Set the components of T
    float T1 = sinf(2*PI*p/dimin.x);
    float T2 = sinf(2*PI*q/dimin.y);
    float T3 = sinf(2*PI*r/dimin.z);
    // Set the denominator
    float denom = 1.0 - beta * (d11 * T1 * T1 + d22 * T2 * T2 + d33 * T3 * T3);
    // Set the components
    eigenvals[idx + 0*dimcoefs.x*dimcoefs.y*dimcoefs.z] = d11 + beta * d11 * T1 * d11 * T1 / denom; // A(1,1)
    eigenvals[idx + 1*dimcoefs.x*dimcoefs.y*dimcoefs.z] = d22 + beta * d22 * T2 * d22 * T2 / denom; // A(2,2)
    eigenvals[idx + 2*dimcoefs.x*dimcoefs.y*dimcoefs.z] = d33 + beta * d33 * T3 * d33 * T3 / denom; // A(3,3)
    eigenvals[idx + 3*dimcoefs.x*dimcoefs.y*dimcoefs.z] = beta * d11 * T1 * d22 * T2 / denom; // A(1,2)
    eigenvals[idx + 4*dimcoefs.x*dimcoefs.y*dimcoefs.z] = beta * d11 * T1 * d33 * T3 / denom; // A(1,3)
    eigenvals[idx + 5*dimcoefs.x*dimcoefs.y*dimcoefs.z] = beta * d22 * T2 * d33 * T3 / denom; // A(2,3)
}

__global__ void getEigenValsKernelv2(float *eigenvals, float gamma, float mu, float lambda, dim dimcoefs, dim dimin) {
    /// Determine the relative index of the current GPU thread
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int r = blockIdx.y;
    int q = idx / dimcoefs.x;
    int p = idx % dimcoefs.x;
    // Check that the indices are within image bounds
    if ((r < 0) || (q < 0) || (p < 0) ||
        (r > dimcoefs.z - 1) || (q > dimcoefs.y - 1) || (p > dimcoefs.x - 1)) {
        return;
    }
    // Compute the absolute index for future use
    idx = r + q*dimcoefs.z + p*dimcoefs.z*dimcoefs.y;
    // Get the center of the grid
    int centerx = floorf(dimin.x/2)+1;
    int centery = floorf(dimin.y/2)+1;
    int centerz = floorf(dimin.z/2)+1;

    if (p > centerx) {p = centerx - p;}
    if (q > centery) {q = centery - q;}

    // Get the norm of the frequency
    float normpsq = powf((float) p/ (float) dimin.x, 2.0) + powf((float) q/(float) dimin.y, 2.0) + powf((float) r/ (float) dimin.z, 2.0);
    // Set beta
    float denom1 = (float) 1.0 / (float) (1.0 + 4.0*PI*PI*gamma*mu*normpsq);
    float denom2 = (float) 1.0 / (float)  (1.0 + 4.0*PI*PI*gamma*(2*mu+lambda)*normpsq);
    // Set the eigenvals
    eigenvals[idx + 0*dimcoefs.x*dimcoefs.y*dimcoefs.z] = denom1 - denom2 * p * p / (dimin.x * dimin.x); // A(1,1)
    eigenvals[idx + 1*dimcoefs.x*dimcoefs.y*dimcoefs.z] = denom1 - denom2 * q * q / (dimin.y * dimin.y); // A(2,2)
    eigenvals[idx + 2*dimcoefs.x*dimcoefs.y*dimcoefs.z] = denom1 - denom2 * r * r / (dimin.z * dimin.z); // A(3,3)
    eigenvals[idx + 3*dimcoefs.x*dimcoefs.y*dimcoefs.z] = -denom2 * p * q / (dimin.x * dimin.y); // A(1,2)
    eigenvals[idx + 4*dimcoefs.x*dimcoefs.y*dimcoefs.z] = -denom2 * p * r / (dimin.x * dimin.z); // A(1,3)
    eigenvals[idx + 5*dimcoefs.x*dimcoefs.y*dimcoefs.z] = -denom2 * q * r / (dimin.y * dimin.z); // A(2,3)
}

__host__ void getEigenVals(float *eigenvals, float gamma, float mu, float lambda, dim dimcoefs, dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimcoefs.x*dimcoefs.y/numThreads + (((dimcoefs.x*dimcoefs.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimcoefs.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Execute kernel
    getEigenValsKernel <<<nBlocks, threadsPerBlock>>> (eigenvals, gamma, mu, lambda, dimcoefs, dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch update kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    //Done
    return;
}