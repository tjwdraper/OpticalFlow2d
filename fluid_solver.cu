#include "hip/hip_runtime.h"
/*
    We provide the definitions of the functions for image registration using optical flow + fluid 
    regularization. Each CUDA kernel is wrapped around a host function.

    @company: UMC Utrecht
    @author: Tom Draper
    @date: 15-6-2022
*/
#include <hip/hip_runtime.h>
#include <mex.h>
#include <dim.cuh>
#include <coord3D.cuh>
#include <fluid_solver.cuh>
#include <gradients.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include <mex.h>

struct norm_unary : public unary_function<coord3D, coord3D> {
    __host__ __device__ float operator()(const coord3D &c) const {
        return sqrt(c.x*c.x+c.y*c.y+c.z*c.z);
    }
};

/*
    Jacobian
*/

__global__ void jacobian_determinantKernel(float *jacobian, coord3D *motion, dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) || 
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    //
    coord3D dudx = partial_x(motion, idx, i, dimin);
    coord3D dudy = partial_y(motion, idx, j, dimin);
    coord3D dudz = partial_z(motion, idx, k, dimin);
    // Get the minimum value of the Jacobian of the transformation
    jacobian[idx] = (1.0+dudx.x)*((1.0+dudy.y)*(1.0+dudz.z)-dudy.z*dudz.y) - 
            dudx.y*(dudy.x*(1.0+dudz.z)-dudz.x*dudy.z) + 
            dudx.z*(dudy.x*dudz.y-dudz.x*(1.0+dudy.y));
    // Done 
    return;
}

__host__ void jacobian_determinant(float *jacobian, coord3D *motion, dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    jacobian_determinantKernel <<<nBlocks, threadsPerBlock>>> (jacobian,
            motion,
            dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch jacobian kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    // Check if regridding procedure should be initiated
    return;
}

/*
    Dirichlet boundary conditions
*/
__global__ void enforceDirichletBoundaryConditionsKernel(coord3D *field, dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Check if thread calculates on boundary
    if ((k == 0) || (j == 0) || (i == 0) || 
        (k == dimin.z-1) || (j == dimin.y-1) || (i == dimin.x-1)) {
        // Compute the absolute index for future use.
        idx = k * dimin.x * dimin.y + j * dimin.x + i;
        // Set field to zero
        field[idx] = 0.0;
    }
}

__host__ void enforceDirichletBoundaryConditions(coord3D *field, dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    enforceDirichletBoundaryConditionsKernel <<< nBlocks, threadsPerBlock >>> (field, dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch enforceDirichletBoundaryConditions kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    // Check if regridding procedure should be initiated
    return;
}

/* 
    Neumann boundary conditions
*/

__global__ void enforceNeumannBoundaryConditionsKernel(coord3D *field, dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    // Get the step
    dim step(1, dimin.x, dimin.x * dimin.y);
    // Sides
    if (i == 0)              { field[idx] = field[idx + step.x]; }
    if (i == dimin.x - 1)    { field[idx] = field[idx - step.x]; }
    if (j == 0)              { field[idx] = field[idx + step.y]; }
    if (j == dimin.y - 1)    { field[idx] = field[idx - step.y]; }
    if (k == 0)              { field[idx] = field[idx + step.z]; } 
    if (k == dimin.z - 1)    { field[idx] = field[idx - step.z]; }
    // Vertices
    if ((i == 0) && (j == 0))                    { field[idx] = field[idx + step.x + step.y]; }
    if ((i == 0) && (j == dimin.y-1))            { field[idx] = field[idx + step.x - step.y]; }
    if ((i == 0) && (k == 0))                    { field[idx] = field[idx + step.x + step.z]; }
    if ((i == 0) && (k == dimin.z-1))            { field[idx] = field[idx + step.x - step.z]; }
    if ((i == dimin.x-1) && (j == 0))            { field[idx] = field[idx - step.x + step.y]; }
    if ((i == dimin.x-1) && (j == dimin.y-1))    { field[idx] = field[idx - step.x - step.y]; }
    if ((i == dimin.x-1) && (k == 0))            { field[idx] = field[idx - step.x + step.z]; }
    if ((i == dimin.x-1) && (k == dimin.z-1))    { field[idx] = field[idx - step.x - step.z]; }
    if ((j == 0) && (k == 0))                    { field[idx] = field[idx + step.y + step.z]; }
    if ((j == 0) && (k == dimin.z-1))            { field[idx] = field[idx + step.y - step.z]; }
    if ((j == dimin.y-1) && (k == 0))            { field[idx] = field[idx - step.y + step.z]; }
    if ((j == dimin.y-1) && (k == dimin.z-1))    { field[idx] = field[idx - step.y - step.z]; }
    // Corners
    if ((i == 0) && (j == 0) && (k == 0))                            {field[idx] = field[idx + step.x + step.y + step.z];}
    if ((i == 0) && (j == 0) && (k == dimin.z-1))                    {field[idx] = field[idx + step.x + step.y - step.z];}
    if ((i == 0) && (j == dimin.y-1) && (k == 0))                    {field[idx] = field[idx + step.x - step.y + step.z];}
    if ((i == 0) && (j == dimin.y-1) && (k == dimin.z-1))            {field[idx] = field[idx + step.x - step.y - step.z];}
    if ((i == dimin.x-1) && (j == 0) && (k == 0))                    {field[idx] = field[idx - step.x + step.y + step.z];}
    if ((i == dimin.x-1) && (j == 0) && (k == dimin.z-1))            {field[idx] = field[idx - step.x + step.y - step.z];}
    if ((i == dimin.x-1) && (j == dimin.y-1) && (k == 0))            {field[idx] = field[idx - step.x - step.y + step.z];}
    if ((i == dimin.x-1) && (j == dimin.y-1) && (k == dimin.z-1))    {field[idx] = field[idx - step.x - step.y - step.z];}
    // Done
    return;
}

__host__ void enforceNeumannBoundaryConditions(coord3D *field, dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    enforceNeumannBoundaryConditionsKernel <<< nBlocks, threadsPerBlock >>> (field, dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch enforceNeumannBoundaryConditions kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    // Check if regridding procedure should be initiated
    return;
}

/*
    Get force vector from SSD

*/
__global__ void generate_forceKernel(coord3D *force, 
    float *Iref, float *Ireg, 
    dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    force[idx] = coord3D(partial_x(Ireg, idx, i, dimin),
                         partial_y(Ireg, idx, j, dimin),
                         partial_z(Ireg, idx, k, dimin)) * (Ireg[idx] - Iref[idx]);
    // Done 
    return;
}

__host__ void generate_force(coord3D *force, 
    float *Iref, float *Ireg, 
    dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    generate_forceKernel <<<nBlocks, threadsPerBlock>>> (force,
            Iref, Ireg,
            dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch generate_force kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    // Check if regridding procedure should be initiated
    return;
}

__global__ void generate_add_reverse_forceKernel(coord3D *force, 
    float *Imov, float *Ireg, 
    dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;   
    
    coord3D reverse_force = coord3D(partial_x(Ireg, idx, i, dimin),
                                    partial_y(Ireg, idx, j, dimin),
                                    partial_z(Ireg, idx, k, dimin)) * (Imov[idx] - Ireg[idx]);
    force[idx] = (force[idx] + reverse_force)/2;
    // Done 
    return;
}

__host__ void generate_add_reverse_force(coord3D *force, 
    float *Imov, 
    float *Ireg, 
    dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    generate_add_reverse_forceKernel <<<nBlocks, threadsPerBlock>>> (force,
            Imov, Ireg,
            dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch generate_force kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    // Check if regridding procedure should be initiated
    return;
}

__global__ void generate_gradient_forceKernel(coord3D *force, 
    float *Iref, float *Ireg, 
    dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    //
    float eta = 0.03;
    coord3D grad_f = coord3D(partial_x(Ireg, idx, i, dimin),
                             partial_y(Ireg, idx, j, dimin),
                             partial_z(Ireg, idx, k, dimin));
    coord3D grad_g = coord3D(partial_x(Iref, idx, i, dimin),
                             partial_y(Iref, idx, j, dimin),
                             partial_z(Iref, idx, k, dimin));
    float norm_eta_f = sqrt(grad_f.x*grad_f.x + grad_f.y*grad_f.y + grad_f.z*grad_f.z + eta*eta);
    float norm_eta_g = sqrt(grad_g.x*grad_g.x + grad_g.y*grad_g.y + grad_g.z*grad_g.z + eta*eta);
    //
    grad_f /= norm_eta_f;
    grad_g /= norm_eta_g;
    //
    float innerprod = grad_f.x*grad_g.x + grad_f.y*grad_g.y + grad_f.z*grad_g.z;
    //
    float dfdxx = partial_xx(Ireg, idx, i, dimin);
    float dfdyy = partial_yy(Ireg, idx, j, dimin);
    float dfdzz = partial_zz(Ireg, idx, k, dimin);
    float dfdxy = partial_xy(Ireg, idx, i, j, dimin);
    float dfdxz = partial_xz(Ireg, idx, i, k, dimin);
    float dfdyz = partial_yz(Ireg, idx, j, k, dimin);
    //
    force[idx].x = (float)- 2.0*innerprod/norm_eta_f*( dfdxx*(grad_g.x - innerprod*grad_f.x) +
                                             dfdxy*(grad_g.y - innerprod*grad_f.y) +
                                             dfdxz*(grad_g.z - innerprod*grad_f.z) );
    force[idx].x = (float) -2.0*innerprod/norm_eta_f*( dfdxy*(grad_g.x - innerprod*grad_f.x) +
                                             dfdyy*(grad_g.y - innerprod*grad_f.y) +
                                             dfdyz*(grad_g.z - innerprod*grad_f.z) );
    force[idx].x = (float) -2.0*innerprod/norm_eta_f*( dfdxz*(grad_g.x - innerprod*grad_f.x) +
                                             dfdyz*(grad_g.y - innerprod*grad_f.y) +
                                             dfdzz*(grad_g.z - innerprod*grad_f.z) );

    // Done 
    return;
}

__host__ void generate_gradient_force(coord3D *force, 
    float *Iref, float *Ireg, 
    dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    generate_gradient_forceKernel <<<nBlocks, threadsPerBlock>>> (force,
            Iref, Ireg,
            dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch generate_force kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    // Check if regridding procedure should be initiated
    return;
}

__global__ void optFlowKernel(coord3D *forceField,
    float *refImage, float *movImage,
    coord3D *motion,
    dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    forceField[idx] = coord3D(partial_x(movImage, idx, i, dimin),
                              partial_y(movImage, idx, j, dimin),
                              partial_z(movImage, idx, k, dimin));
    forceField[idx] *= movImage[idx] - refImage[idx] + motion[idx].x*forceField[idx].x + motion[idx].y*forceField[idx].y + motion[idx].z*forceField[idx].z; 
    // Done 
    return;
}

__host__ void optFlow(coord3D *forceField, 
    float *refImage, float *movImage,
    coord3D *motion, 
    dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    optFlowKernel <<<nBlocks, threadsPerBlock>>> (forceField,
            refImage, movImage,
            motion,
            dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch optFlow kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    // Check if regridding procedure should be initiated
    return;
}

/*
        Adaptive time-step and increment in motion field
*/
__global__ void incrementKernel(coord3D *increment,
    coord3D *motion,
    coord3D *velocity,
    dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    // 
    coord3D dudx = partial_x(motion, idx, i, dimin);
    coord3D dudy = partial_y(motion, idx, j, dimin);
    coord3D dudz = partial_z(motion, idx, k, dimin);
    // Get values for R and get dt
    increment[idx] = velocity[idx] - dudx*velocity[idx].x - dudy*velocity[idx].y - dudz*velocity[idx].z; 
    // Done
    return;
}

__host__ void increment(coord3D *increment, 
    coord3D *motion,
    coord3D *velocity, 
    dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    incrementKernel <<<nBlocks, threadsPerBlock>>> (increment,
        motion,
        velocity,
        dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch increment kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    //Done
    return; 
}


/*
    Integrate motion field over time using explicit Euler method
*/
__global__ void integrateKernel(coord3D *motion,
    coord3D *increment, float dt,
    dim dimin) {
    // Determine the relative index of the current GPU thread.
    int idx = blockIdx.x *blockDim.x + threadIdx.x;
    int k = blockIdx.y;
    int j = idx / dimin.x;
    int i = idx % dimin.x;
    // Check that the indices are within image bounds.
    if ((k < 0) || (j < 0) || (i < 0) ||
        (k > dimin.z - 1) || (j > dimin.y - 1) || (i > dimin.x - 1)) {
        return;
    }
    // Compute the absolute index for future use.
    idx = k * dimin.x * dimin.y + j * dimin.x + i;
    // Calculate new entry
    motion[idx] += increment[idx]*dt;
    // Done
    return;
}

__host__ void integrate(coord3D *motion, 
    coord3D *increment, float dt,
    dim dimin) {
    // Establish the number of threads per block and the arrangement of the
    // blocks in the grid.
    int numThreads = 256;
    dim3 threadsPerBlock(numThreads, 1, 1);
    int numberOfBlocksX = dimin.x*dimin.y/numThreads + (((dimin.x*dimin.y)%numThreads==0)?0:1);
    int numberOfBlocksY = dimin.z;
    dim3 nBlocks(numberOfBlocksX, numberOfBlocksY, 1);
    // Launch kernel.
    integrateKernel <<<nBlocks, threadsPerBlock>>> (motion, increment, dt, dimin);
    // Sync
    hipDeviceSynchronize();
    // Check for errors.
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        mexPrintf("Failed to launch eulerIntegration kernel (error code: %s)!\n",
            hipGetErrorString(err));
    }
    //Done
    return;
}